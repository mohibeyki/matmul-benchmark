#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <cstdlib>
#include <cstdio>
#include <ctime>

#define BLOCK_SIZE 16

#ifndef DEVICE_COUNT
#define DEVICE_COUNT 1
#endif

void print_matrix_2D(float *A, int nr_rows_A, int nr_cols_A)
{
        for (int i = 0; i < nr_rows_A; ++i)
        {
                for (int j = 0; j < nr_cols_A; ++j)
                        printf("%.0f ", A[i * nr_cols_A + j]);
                printf("\n");
        }
        printf("\n");
}

__global__ void gpu_matrix_mult(float *a, float *b, float *c, int ms, int me, int n, int k)
{
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        float sum = 0;
        if (col < k && row >= ms && row < me)
        {
                for (int i = 0; i < n; i++)
                        sum += a[row * n + i] * b[i * k + col];
                c[row * k + col] = sum;
        }
}

int main(int argc, char *argv[])
{
        if (argc < 4)
        {
                printf("use ./gsmm_cuda m n k\n");
                return 1;
        }
        int m = atoi(argv[1]), n = atoi(argv[2]), k = atoi(argv[3]);

        float *a = new float[m * n];
        float *b = new float[n * k];
        float *c = new float[m * k];
        int *min_rows = new int[m];

        srand(time(0));
        for (int i = 0; i < m; i++)
                for (int j = 0; j < n; j++)
                        a[i * n + j] = rand() % 2 + 1;

        for (int i = 0; i < n; i++)
                for (int j = 0; j < k; j++)
                        b[i * k + j] = rand() % 2 + 1;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        float *d_a[DEVICE_COUNT], *d_b[DEVICE_COUNT], *d_c[DEVICE_COUNT];
        int *d_min_rows[DEVICE_COUNT];

        for (int i = 0; i < DEVICE_COUNT; i++)
        {
                hipSetDevice(i);
                hipMalloc((void **)&d_a[i], sizeof(float) * m * n);
                hipMalloc((void **)&d_b[i], sizeof(float) * n * k);
                hipMalloc((void **)&d_c[i], sizeof(float) * m * k);
                hipMalloc((void **)&d_min_rows[i], sizeof(int) * m);
                hipMemcpy(d_a[i], a, sizeof(float) * m * n, hipMemcpyHostToDevice);
                hipMemcpy(d_b[i], b, sizeof(float) * n * k, hipMemcpyHostToDevice);
                hipMemcpy(d_min_rows[i], min_rows, sizeof(int) * m, hipMemcpyHostToDevice);
        }

        unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
        unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;

        dim3 numBlocks(grid_cols, grid_rows);
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);

        for (int i = 0; i < DEVICE_COUNT; i++)
        {
                hipSetDevice(i);
                int ms = m / DEVICE_COUNT * i, me = m / DEVICE_COUNT * (i + 1);
                int device_mem = m * k / DEVICE_COUNT;
                hipEventRecord(start, 0);
                gpu_matrix_mult<<<numBlocks, blockSize>>>(d_a[i], d_b[i], d_c[i], ms, me, n, k);
                hipMemcpy(c + (device_mem * i), d_c[i] + (device_mem * i), sizeof(float) * device_mem, hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                hipEventRecord(stop, 0);
                hipEventSynchronize(stop);
                float gpu_elapsed_time_ms;
                hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
                printf("%f\n", gpu_elapsed_time_ms);
        }

        return 0;
}
